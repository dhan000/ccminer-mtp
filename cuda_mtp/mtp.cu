#include "hip/hip_runtime.h"


#include "argon2ref/argon2.h"
#include "merkletree/mtp.h"

#include <unistd.h>
#include "miner.h"
#include "cuda_helper.h"
#define memcost 4*1024*1024

extern void mtp_cpu_init(int thr_id, uint32_t threads);

extern uint32_t mtp_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNounce);

extern void mtp_setBlockTarget(int thr_id,const void* pDataIn, const void *pTargetIn, const void * zElement);
extern void mtp_fill(uint32_t d, const uint64_t *Block, uint32_t offset, uint32_t datachunk);
extern void mtp_fill_1b(int thr_id,uint64_t *Block, uint32_t block_nr);
extern void mtp_i_cpu(int thr_id, uint32_t *block_header);
extern uint8_t *mtp_init3(argon2_instance_t *instance, int thr_id);

#define HASHLEN 32
#define SALTLEN 16
#define PWD "password"
//#define MTP_L 64

static bool init[MAX_GPUS] = { 0 };
static __thread uint32_t throughput = 0;
static uint32_t JobId[MAX_GPUS] = {0};
//static  MerkleTree::Elements TheElements[MAX_GPUS];
static  MerkleTree ordered_tree[MAX_GPUS];
static  unsigned char TheMerkleRoot[MAX_GPUS][16];
static  argon2_context context[MAX_GPUS];
static argon2_instance_t instance[MAX_GPUS];


static std::vector<uint8_t*> MEM[MAX_GPUS];


//static pthread_mutex_t work_lock;
//static pthread_barrier_t barrier;
extern "C" int scanhash_mtp(int nthreads, int thr_id, struct work* work, uint32_t max_nonce, unsigned long *hashes_done, struct mtp* mtp)
{
//	if (work_restart[thr_id].restart) return 0;
//	unsigned char TheMerkleRoot[16];
	unsigned char mtpHashValue[32];

//	pthread_mutex_init(&work_lock, NULL);
//	pthread_barrier_init(&barrier, NULL, 1);
int real_maxnonce = UINT32_MAX / nthreads * (thr_id + 1);

//	MerkleTree::Elements TheElements; // = new MerkleTree;
//printf("the job_id from mtp %s\n",work->job_id+8);
	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;
	const uint32_t first_nonce = pdata[19];
	int dev_id = device_map[thr_id];;
	if (opt_benchmark)
		ptarget[7] = 0x00ff;

		uint32_t diff = 5;
		uint32_t TheNonce;

	if (!init[thr_id])
	{

		hipSetDevice(dev_id);
		
		hipDeviceReset();
		hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
//		hipSetDeviceFlags(hipDeviceScheduleYield);

//		int intensity = (device_sm[dev_id] >= 500 && !is_windows()) ? 17 : 16;
//		if (device_sm[device_map[thr_id]] == 500) intensity = 15;
		int intensity = 19;
		throughput = cuda_default_throughput(thr_id, 1U << intensity); // 18=256*256*4;
//		throughput =  1024*64;
		if (init[thr_id]) throughput = min(throughput, max_nonce - first_nonce);

		hipDeviceProp_t props;
		hipGetDeviceProperties(&props, dev_id);


		gpulog(LOG_INFO, thr_id, "Intensity set to %g, %u cuda threads", throughput2intensity(throughput), throughput);


		mtp_cpu_init(thr_id, throughput);

		init[thr_id] = true;

	}

	uint32_t _ALIGN(128) endiandata[20];
	((uint32_t*)pdata)[19] = (pdata[20]); // */0x00100000; // mtp version not the actual nonce
//	((uint32_t*)pdata)[19] = 0x1000;

	for (int k = 0; k < 20; k++) 
		endiandata[k] = pdata[k];
	
//	argon2_context context = init_argon2d_param((const char*)endiandata);
//	argon2_instance_t instance;
//	argon2_ctx_from_mtp(&context, &instance);
//printf("coming here\n");


//pthread_mutex_lock(&work_lock);

if (JobId[thr_id]!= work->data[17]){

gpulog(LOG_WARNING, thr_id, "filling memory");
//restart_threads();
//pthread_barrier_wait(&barrier);
if (JobId[thr_id]!=0)
	free_memory(&context[thr_id], (unsigned char *)instance[thr_id].memory, instance[thr_id].memory_blocks, sizeof(block));

//printf("coming here2\n");
	context[thr_id] = init_argon2d_param((const char*)endiandata);
	argon2_ctx_from_mtp(&context[thr_id], &instance[thr_id]);

for(int i=0;i<MEM[thr_id].size();i++)
	free(MEM[thr_id][i]);
//printf("filling memory\n");
//gpulog(LOG_WARNING, thr_id, "filled first blocks on cpu\n");
mtp_fill_1b(thr_id,instance[thr_id].memory[0 + 0].v,0 + 0);
mtp_fill_1b(thr_id,instance[thr_id].memory[0 + 1].v,0 + 1);

mtp_fill_1b(thr_id,instance[thr_id].memory[2 + 0].v,1048576 + 0);
mtp_fill_1b(thr_id,instance[thr_id].memory[2 + 1].v,1048576 + 1);
mtp_fill_1b(thr_id,instance[thr_id].memory[4 + 0].v,2097152 + 0);
mtp_fill_1b(thr_id,instance[thr_id].memory[4 + 1].v,2097152 + 1);
mtp_fill_1b(thr_id,instance[thr_id].memory[6 + 0].v,3145728 + 0);
mtp_fill_1b(thr_id,instance[thr_id].memory[6 + 1].v,3145728 + 1);
mtp_i_cpu(thr_id,instance[thr_id].block_header);

//	MerkleTree::Elements TheElements = mtp_init2(&instance[thr_id], thr_id);
	uint8_t * x =  mtp_init3(&instance[thr_id], thr_id);

//	ordered_tree[thr_id] = MerkleTree(TheElements, true);
      ordered_tree[thr_id] = MerkleTree(x, true);
//	gpulog(LOG_WARNING, thr_id, "filled blocks on GPU\n");

//for(;;);
	JobId[thr_id] = work->data[17];

	MerkleTree::Buffer root = ordered_tree[thr_id].getRoot();
//for(;;);
	std::copy(root.begin(), root.end(), TheMerkleRoot[thr_id]);
	MEM[thr_id]=ordered_tree[thr_id].getMem();
//	mtp_setBlockTarget(0,endiandata,ptarget,&TheMerkleRoot);
	mtp_setBlockTarget(thr_id, endiandata, ptarget, &TheMerkleRoot[thr_id]);

//for(int i=0;i<16;i++)printf("%x ",TheMerkleRoot[thr_id][i]);
//printf("\n");

//for(;;);



/*
const int datachunk = 512;
for (int i=0;i<((uint32_t)memcost/ datachunk) ;i++) {
uint64_t *Truc =(uint64_t *) malloc(128* datachunk*sizeof(uint64_t));
	
	for (int j=0;j<datachunk;j++)
		memcpy(&Truc[128*j],instance[thr_id].memory[datachunk*i+j].v,128*sizeof(uint64_t));

	mtp_fill(thr_id,Truc, i, datachunk);
//	mtp_fill(1, Truc, i, datachunk);
	free(Truc);
}
*/
gpulog(LOG_WARNING, thr_id, "memory filled %d chunks",MEM[thr_id].size());
}

//pthread_mutex_unlock(&work_lock);



	if (work_restart[thr_id].restart) goto TheEnd;
		pdata[19] = first_nonce;
//do  
	{
		int order = 0;
		uint32_t foundNonce;

		*hashes_done = pdata[19] - first_nonce + throughput;
//printf("first nonce %08x thr_id %08x\n", pdata[19],thr_id);

		foundNonce = mtp_cpu_hash_32(thr_id, throughput, pdata[19]);

		uint32_t _ALIGN(64) vhash64[8];
		if (foundNonce != UINT32_MAX)
		{

			block_mtpProof TheBlocksAndProofs[140];
			uint256 TheUint256Target[1];
			TheUint256Target[0] = ((uint256*)ptarget)[0];

			blockS nBlockMTP[MTP_L *2];
			unsigned char nProofMTP[MTP_L * 3 * 353 ];
			
			uint32_t is_sol = mtp_solver(thr_id, foundNonce, &instance[thr_id], nBlockMTP,nProofMTP, TheMerkleRoot[thr_id], mtpHashValue, ordered_tree[thr_id], endiandata,TheUint256Target[0]);

			if (is_sol==1 /*&& fulltest(vhash64, ptarget)*/) {
				int res = 1;
				work_set_target_ratio(work, vhash64);		

				pdata[19] =/*swab32*/(foundNonce);

/// fill mtp structure
				mtp->MTPVersion = 0x1000;
			for (int i=0;i<16;i++) 
				mtp->MerkleRoot[i] = TheMerkleRoot[thr_id][i];
			for (int i = 0; i<32; i++)
				mtp->mtpHashValue[i] = mtpHashValue[i];
			
			for (int j=0;j<(MTP_L * 2);j++)
				for (int i=0;i<128;i++)
				mtp->nBlockMTP[j][i]= nBlockMTP[j].v[i];
                int lenMax =0; 
				int len = 0;

				memcpy(mtp->nProofMTP, nProofMTP, sizeof(unsigned char)* MTP_L * 3 * 353);


//				printf("found a solution, nonce %08x\n",pdata[19]);
//				free_memory(&context, (unsigned char *)instance.memory, instance.memory_blocks, sizeof(block));
//				pthread_mutex_destroy(&work_lock);
				return res;

			} else {
				gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU!", foundNonce);
			}
		}
		work_set_target_ratio(work, vhash64);
/*
		if ((uint64_t)throughput + pdata[19] >= max_nonce) {
			pdata[19] = max_nonce;
			break;
		}
*/
		pdata[19] += throughput;

		if (pdata[19]>=real_maxnonce) {
			gpulog(LOG_WARNING, thr_id, "OUT OF NONCE %x >= %x", pdata[19], real_maxnonce);
			abort();
		}

//		be32enc(&endiandata[19], pdata[19]);
	}   
	//while (!work_restart[thr_id].restart && pdata[19]<0xffffffff && pdata[19]<real_maxnonce);


TheEnd:
//	free_memory(&context, (unsigned char *)instance.memory, instance.memory_blocks, sizeof(block));
	*hashes_done = pdata[19] - first_nonce;

//	ordered_tree.~MerkleTree();
//	TheElements.clear();
//	pthread_mutex_destroy(&work_lock);
	return 0;
}


